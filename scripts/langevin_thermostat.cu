#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "system.h"
#include "cuda_opt_constants.h"
#include "Particle.h"

// For the langevin thermostat
__global__ void setupRandomStates(hiprandState* states, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

float CalculateCurrentTemperature(Particle *particles, int numParticles) {
    float totalKineticEnergy = 0.0;

    for (int i = 0; i < numParticles; ++i) {
        totalKineticEnergy += particles[i].GetKineticEnergy();
    }

    float temperature = (2.0 / (3.0 * numParticles * kB)) * totalKineticEnergy;
    return temperature;
}

__global__ void applyLangevinThermostat(Particle* particles, int N, float dt, float gamma, float temperature, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        // Generate a normally distributed random number
        float randX = hiprand_normal_double(&states[idx]);
        float randY = hiprand_normal_double(&states[idx]);
        float randZ = hiprand_normal_double(&states[idx]);

        // Calculate the random force magnitude
        float randomForceMagnitude = sqrt(2.0 * particles[idx].mass * kB_d * temperature * gamma * dt);

        // Update velocity
        particles[idx].velocityX += (particles[idx].forceX / particles[idx].mass - gamma * particles[idx].velocityX) * dt 
                                   + randomForceMagnitude * randX / particles[idx].mass;
        particles[idx].velocityY += (particles[idx].forceY / particles[idx].mass - gamma * particles[idx].velocityY) * dt 
                                   + randomForceMagnitude * randY / particles[idx].mass;
        particles[idx].velocityZ += (particles[idx].forceZ / particles[idx].mass - gamma * particles[idx].velocityZ) * dt 
                                   + randomForceMagnitude * randZ / particles[idx].mass;
    }
}