#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <array>
#include <cmath>
#include <omp.h>

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "Random64.h"
#include "Particle.h"
#include "Collider.h"

#include "system.h"
#include "write_settings.h"
#include "cuda_opt_constants.h"


int main() {

    // Read input parameters
    Readdat();

    if (L / 2 < cutoff) {
        cerr << "Error : The cutoff distance is greater than half the box dimension L / 2 :" << L / 2 << " - Rc : " << cutoff << endl;
        return 1;
    }
/*    
    cout << "dt                     : " << dt << endl;
    cout << "equilibrationSteps     : " << equilibrationSteps << endl;
    cout << "NumberOfSteps          : " << NumberOfSteps << endl;
    cout << "kB                     : " << kB << endl;
    cout << "epsilon                : " << epsilon << endl;
    cout << "sigma                  : " << sigma << endl;
    cout << "cutoff                 : " << cutoff << endl;
    cout << "Gamma                  : " << Gamma << endl;
    cout << "T_desired              : " << T_desired << endl;
    cout << "defaultMass            : " << defaultMass << endl;
    cout << "InitialVelocity        : " << InitialVelocity << endl;
    cout << "RHO                    : " << RHO << endl;
    cout << "L                      : " << L << endl;
    cout << "sigma_6                : " << sigma_6 << endl;
    cout << "forceNormalCutOff      : " << forceNormalCutOff << endl;
    cout << "epsilon_sigma_6        :   " << epsilon_sigma_6 << endl;
    cout << "potentialEnergy_cutoff : " << potentialEnergy_cutoff << endl;*/

    // Write setup
    OutputManager outputManager;
    outputManager.setOutputNames(simulationLabel);
    outputManager.openFiles();
    std::ofstream& outFile_positions  = outputManager.getPosFile();
    std::ofstream& outFile_velocities = outputManager.getVelFile();
    std::ofstream& outFile_forces     = outputManager.getForcesFile();
    std::ofstream& outFile_rdf  = outputManager.getRdfFile();
    std::ofstream& outFile_vacf = outputManager.getVafFile();
    std::ofstream& outFile_msd  = outputManager.getMsdFile();
    std::ofstream& outFile_temperature = outputManager.getTemperatureFile();

    // Misc
    float time, radius, kineticEnergy, potentialEnergy, T_current;
    int i, drawTime, currentTimeStep;

    // Intit collider
    Collider collider;
    collider.Init();

    // Init partciles 
    Particle particles[N];
    Crandom randomGenerator(0);

    // Set initial velocities and positions in a fcc config
    int unitCellsPerSide = std::cbrt(N / 4);
    float a = Lx / unitCellsPerSide;
    //float a = 1.6823909;

    cout << "lattice constant       : " << a <<  endl;
    cout << "unit cell per side     : " << unitCellsPerSide <<  endl;

    std::vector<std::array<double, 3>> velocities(N);
    std::vector<std::array<double, 3>> positions(N); // Store initial positions

    // Assign random velocities and positions
    float totalVx = 0, totalVy = 0, totalVz = 0;
    int particleIndex = 0;
    for (int ix = 0; ix < unitCellsPerSide; ix++) {
        for (int iy = 0; iy < unitCellsPerSide; iy++) {
            for (int iz = 0; iz < unitCellsPerSide; iz++) {
                std::vector<std::array<double, 3>> unitCellPositions = {
                    {ix * a, iy * a, iz * a},
                    {(ix + 0.5) * a, (iy + 0.5) * a, iz * a},
                    {ix * a, (iy + 0.5) * a, (iz + 0.5) * a},
                    {(ix + 0.5) * a, iy * a, (iz + 0.5) * a}
                };

                for (auto& pos : unitCellPositions) {
                    if (particleIndex < N) {
                        float theta = 2 * M_PI * randomGenerator.r();
                        float phi = acos(2 * randomGenerator.r() - 1);
                        float randomInitialVelocity = randomGenerator.r() * InitialVelocity;

                        float velocityX0 = randomInitialVelocity * sin(phi) * cos(theta);
                        float velocityY0 = randomInitialVelocity * sin(phi) * sin(theta);
                        float velocityZ0 = randomInitialVelocity * cos(phi);

                        velocities[particleIndex] = {velocityX0, velocityY0, velocityZ0};
                        positions[particleIndex] = pos;

                        totalVx += velocityX0;
                        totalVy += velocityY0;
                        totalVz += velocityZ0;

                        particleIndex++;
                    }
                }
            }
        }
    }

    // Zero initial momenta 
    float avgVx = totalVx / N;
    float avgVy = totalVy / N;
    float avgVz = totalVz / N;

    for (int i = 0; i < N; i++) {
        particles[i].Init(
            positions[i][0], positions[i][1], positions[i][2],
            velocities[i][0] - avgVx, velocities[i][1] - avgVy, velocities[i][2] - avgVz,
            defaultMass, radius);
    }

    // ***************************************************************
    // ******* CUDA stuff
    // ***************************************************************
    // kernel config
    int blockSize = 256; 
    int numBlocks = (N + blockSize - 1) / blockSize;

    // Device memory alloc
    Particle* dev_particles;
    hipMalloc(&dev_particles, N * sizeof(Particle));
    hipMemcpy(dev_particles, particles, N * sizeof(Particle), hipMemcpyHostToDevice);

    float* dev_partialPotentialEnergy;
    hipMalloc(&dev_partialPotentialEnergy, N * sizeof(float));

    // Random states
    hiprandState* devStates;
    hipMalloc(&devStates, N * sizeof(hiprandState));
    setupRandomStates<<<numBlocks, blockSize>>>(devStates, 1242547);

/* 
    // Stress tensor 
    float* dev_stressTensor;
    hipMalloc(&dev_stressTensor, 9 * sizeof(float)); // 3x3 stress tensor
    hipMemset(dev_stressTensor, 0, 9 * sizeof(float));

    float boxVolume = L*L*L;
    calculateStressTensorCUDA<<<numBlocks, blockSize>>>(dev_particles, dev_stressTensor, N, boxVolume);
    float stressTensor[9];
    hipMemcpy(stressTensor, dev_stressTensor, 9 * sizeof(float), hipMemcpyDeviceToHost);*/

    //////////////////////////////////////////////////////////////////////////////
    //// Neighbor list
    //////////////////////////////////////////////////////////////////////////////
    // displacement setup
    float displacementThreshold = displacementProportion;

    float zero = 0.0f;
    float maxDisplacement = zero;
    float* dev_maxDisplacement;
    hipMalloc(&dev_maxDisplacement, sizeof(float));
    hipMemcpy(dev_maxDisplacement, &zero, sizeof(float), hipMemcpyHostToDevice);

    // Cell list setup
    //// ** We need to upgrade to consider Lx, Ly, Lz different
    float cellLength = Lx / skin; 

    int numCellsX = static_cast<int>(floor(Lx / cellLength));
    int numCellsY = static_cast<int>(floor(Ly / cellLength));
    int numCellsZ = static_cast<int>(floor(Lz / cellLength));

    // Alloc mem in GPU for the cells
    int totalNumCells = numCellsX * numCellsY * numCellsZ;
    Cell* dev_cells;
    hipMalloc(&dev_cells, totalNumCells * sizeof(Cell));

    float cellSize = Lx / numCellsX;
    cout << "cellSize : " << cellSize << " " << cellSize *  numCellsX << endl;

    //////////////////////////////////////////////////////////////////////////////
    //// Equilibartion
    //////////////////////////////////////////////////////////////////////////////
    generateStencils<<<numBlocks, blockSize>>>(dev_cells, numCellsX, numCellsY, numCellsZ, cellSize, cutoff);
    resetCells<<<numBlocks, blockSize>>>(dev_cells, totalNumCells);
    assignParticlesToCells<<<numBlocks, blockSize>>>(dev_particles, dev_cells, N, cellSize, numCellsX, numCellsY, numCellsZ, Lx, Ly, Lz);
/*    
    hipMemcpy(cells, dev_cells, totalNumCells * sizeof(Cell), hipMemcpyDeviceToHost);
    cout << " Cell dim : " << numCellsX << " " << numCellsY << " " << numCellsZ << " "<< endl;
    for(int mm = 0; mm < totalNumCells; mm++){
        cout << " **** Number of particles per cell : " << cells[mm].numParticles << endl;
        cout << mm << " ----------------------------" << endl;
        if(cells[mm].numParticles != 0){
            for(int nn = 0; nn < cells[mm].numParticles; nn++){
                cout << cells[mm].particleIndices[nn] << " ";
            }
        }
        cout << endl;
        cout << " **** Number of stencils per cell : " << cells[mm].stencilSize << endl;
        if(cells[mm].stencilSize == 0){
            cout << mm << "**************** Stencil ***************" << endl;
            for(int nn = 0; nn < cells[mm].stencilSize; nn++){
                cout << cells[mm].halfstencil[nn].x << " " << cells[mm].halfstencil[nn].y << " "<< cells[mm].halfstencil[nn].z << endl;
            }
        }
        cout << endl;
    }
    hipMemcpy(particles, dev_particles, N * sizeof(Particle), hipMemcpyDeviceToHost);
    for(int mm = 0; mm < N; mm++){
        cout << " Number of neighbors per particle : " << particles[mm].numNeighbors << endl;
        cout << mm << " ----------------------------" << endl;
        if(particles[mm].numNeighbors != 0){
            for(int nn = 0; nn < particles[mm].numNeighbors; nn++){
                cout << particles[mm].neighbors[nn] << " ";
            }
        }
        cout << endl;
    }
*/
    updateVerletListKernel<<<numBlocks, blockSize>>>(dev_particles, dev_cells, N, extendedCutoff, numCellsX, numCellsY, numCellsZ, Lx, Ly, Lz);
    collider.CalculateForces(dev_particles, dev_partialPotentialEnergy, N, Lx, Ly, Lz, totalNumCells, dev_cells);
    for (int eqStep = 0; eqStep < equilibrationSteps; eqStep++) {
        if(eqStep % eqVerboseFrame == 0){
            cout << "Equilibration step : " << eqStep << endl;
        }

        // Update particle velocities (half-step)
        updateVelocitiesKernel<<<numBlocks, blockSize>>>(dev_particles, N, dt * 0.5);
        applyLangevinThermostat<<<numBlocks, blockSize>>>(dev_particles, N, dt * 0.5, kB, Gamma, T_desired, devStates);

        // Move particles and calculate displacements
        moveParticlesKernel<<<numBlocks, blockSize>>>(dev_particles, N, dt, Lx, Ly, Lz, dev_maxDisplacement);
        hipMemcpy(&maxDisplacement, dev_maxDisplacement, sizeof(float), hipMemcpyDeviceToHost);

        resetCells<<<numBlocks, blockSize>>>(dev_cells, totalNumCells);
        hipDeviceSynchronize();

        // Update the Verlet list if necessary
        //cout << maxDisplacement << endl;
        if (maxDisplacement > displacementThreshold) {
            assignParticlesToCells<<<numBlocks, blockSize>>>(dev_particles, dev_cells, N, cellSize, numCellsX, numCellsY, numCellsZ, Lx, Ly, Lz);
            updateVerletListKernel<<<numBlocks, blockSize>>>(dev_particles, dev_cells, N, extendedCutoff, numCellsX, numCellsY, numCellsZ, Lx, Ly, Lz);
            hipMemcpy(dev_maxDisplacement, &zero, sizeof(float), hipMemcpyHostToDevice);
        }

        // Calculate forces
        collider.CalculateForces(dev_particles, dev_partialPotentialEnergy, N, Lx, Ly, Lz, totalNumCells, dev_cells);

        // Update particle velocities (half-step)
        updateVelocitiesKernel<<<numBlocks, blockSize>>>(dev_particles, N, dt * 0.5);
        applyLangevinThermostat<<<numBlocks, blockSize>>>(dev_particles, N, dt * 0.5, kB, Gamma, T_desired, devStates);
    }

    cout << "**** Equilibration ended *****" << endl;
    // ***************************************
    // ******* Wrtie eq config ***************
    // ***************************************
    hipMemcpy(particles, dev_particles, N * sizeof(Particle), hipMemcpyDeviceToHost);
    for (i = 0; i < N; i++){
        outFile_positions << i << " " << time << " " << particles[i].GetX() << " " << particles[i].GetY() << " " << particles[i].GetZ() << endl;
        outFile_velocities << i << " " << time << " " << particles[i].GetVelocityX() << " " << particles[i].GetVelocityY() << " " << particles[i].GetVelocityZ() << endl;
        outFile_forces << i << " " << time << " " << particles[i].GetForceX() << " " << particles[i].GetForceY() << " " << particles[i].GetForceZ() << endl;
    }

    // ***************************************
    // ******* Wrtie misc stuff **************
    // ***************************************
    std::vector<float> vacf(maxVACFCount, 0.0);
    int vacfCount = 0;
    int vacfSamplingCount = 0;

    std::vector<float> msd(maxMSDCount, 0.0);
    int msdCount = 0;
    int msdSamplingCount = 0;

    // ******************************
    // ******* MD loop **************
    // ******************************
    for (currentTimeStep = time = drawTime = 0; currentTimeStep < NumberOfSteps; time += dt, drawTime++, currentTimeStep++) {
        /*
        if (1){     
            hipMemcpy(particles, dev_particles, N * sizeof(Particle), hipMemcpyDeviceToHost);
            for (i = 0; i < N; i++){
                outFile_positions << i << " " << time << " " << particles[i].GetX() << " " << particles[i].GetY() << " " << particles[i].GetZ() << endl;
                outFile_velocities << i << " " << time << " " << particles[i].GetVelocityX() << " " << particles[i].GetVelocityY() << " " << particles[i].GetVelocityZ() << endl;
                outFile_forces << i << " " << time << " " << particles[i].GetForceX() << " " << particles[i].GetForceY() << " " << particles[i].GetForceZ() << endl;
            }
        }*/

        if (drawTime % vacf_writeFrame == 0 && vacfSamplingCount < vacfSamplingReps && drawTime > 0) {
            bool shouldWrite = (vacfSamplingCount == vacfSamplingReps - 1 && (vacfCount % (maxVACFCount - 1)) == 0);
            bool shouldReset = (vacfCount % maxVACFCount) == 0;

            if (shouldReset) {
                cout << "Resetting VACF sampling at time: " << time << endl;
                vacfSamplingCount++;
            }

            cout << "Computing VACF for time: " << time << endl;
            hipMemcpy(particles, dev_particles, N * sizeof(Particle), hipMemcpyDeviceToHost);
            ComputeVACF(particles, vacf.data(), N, time, vacfCount, outFile_vacf, shouldReset, shouldWrite);
            vacfCount++;
        }

        if (drawTime % msd_writeFrame == 0 && msdSamplingCount < msdSamplingReps && drawTime > 0) {
            bool shouldWrite = (msdSamplingCount == msdSamplingReps - 1 && (msdCount % (maxMSDCount - 1)) == 0);
            bool shouldReset = (msdCount % maxMSDCount) == 0;

            if (shouldReset) {
                cout << "Resetting MSD sampling at time: " << time << endl;
                msdSamplingCount++;
            }

            cout << "Computing MSD for time: " << time << endl;
            hipMemcpy(particles, dev_particles, N * sizeof(Particle), hipMemcpyDeviceToHost);
            ComputeMSD(particles, msd.data(), N, time, msdCount, outFile_msd, shouldReset, shouldWrite);
            msdCount++;
        }

       if (drawTime % RDF_writeFrame == 0 && drawTime > 0){
            cout << "Writting RDF for time : " << time << endl;
            hipMemcpy(particles, dev_particles, N * sizeof(Particle), hipMemcpyDeviceToHost);
            computeRDFCUDA(particles, N, Lx, Ly, Lz, maxDistance, numBins, time, outFile_rdf);
       }

        if (drawTime % temperature_writeFrame == 0){
            cout << "Writting temperature for time : " << time << endl;
            hipMemcpy(particles, dev_particles, N * sizeof(Particle), hipMemcpyDeviceToHost);
            T_current = CalculateCurrentTemperature(particles, N);
            outFile_temperature << time << " " << T_current << endl;
       }

        // Update particle velocities (half-step)
        updateVelocitiesKernel<<<numBlocks, blockSize>>>(dev_particles, N, dt * 0.5);
        applyLangevinThermostat<<<numBlocks, blockSize>>>(dev_particles, N, dt * 0.5, kB, Gamma, T_desired, devStates);

        // Move particles and calculate displacements
        moveParticlesKernel<<<numBlocks, blockSize>>>(dev_particles, N, dt, Lx, Ly, Lz, dev_maxDisplacement);
        hipMemcpy(&maxDisplacement, dev_maxDisplacement, sizeof(float), hipMemcpyDeviceToHost);

        resetCells<<<numBlocks, blockSize>>>(dev_cells, totalNumCells);
        hipDeviceSynchronize();

        // Update the Verlet list if necessary
        //cout << maxDisplacement << endl;
        if (maxDisplacement > displacementThreshold) {
            assignParticlesToCells<<<numBlocks, blockSize>>>(dev_particles, dev_cells, N, cellSize, numCellsX, numCellsY, numCellsZ, Lx, Ly, Lz);
            updateVerletListKernel<<<numBlocks, blockSize>>>(dev_particles, dev_cells, N, extendedCutoff, numCellsX, numCellsY, numCellsZ, Lx, Ly, Lz);
            hipMemcpy(dev_maxDisplacement, &zero, sizeof(float), hipMemcpyHostToDevice);
        }

        // Calculate forces
        collider.CalculateForces(dev_particles, dev_partialPotentialEnergy, N, Lx, Ly, Lz, totalNumCells, dev_cells);

        // Update particle velocities (half-step)
        updateVelocitiesKernel<<<numBlocks, blockSize>>>(dev_particles, N, dt * 0.5);
        applyLangevinThermostat<<<numBlocks, blockSize>>>(dev_particles, N, dt * 0.5, kB, Gamma, T_desired, devStates);
    }

    hipFree(dev_particles);
    return 0;
}


/* // Write info
if (drawTime % timeFrame == 0) {

    float* partialPotentialEnergy = new float[N];
    hipMemcpy(partialPotentialEnergy, dev_partialPotentialEnergy, N * sizeof(float), hipMemcpyDeviceToHost);
    potentialEnergy = 0;
    for (int i = 0; i < N; i++) {
        potentialEnergy += partialPotentialEnergy[i];
    }
    delete[] partialPotentialEnergy;

    // Get energy
    kineticEnergy = 0;
    for (int i = 0; i < N; i++) kineticEnergy += particles[i].GetKineticEnergy();

    // Get temperature
    T_current = CalculateCurrentTemperature(particles, N);

    // Write
    //outFile_energy << time << " " << kineticEnergy << " " << potentialEnergy << endl;
    outFile_temperature << time << " " << T_current << endl;

    // Assuming the particles array is filled with Particle objects


    // stress tensor cal
    //calculateStressTensorCUDA<<<numBlocks, blockSize>>>(dev_particles, dev_stressTensor, N, boxVolume);
    //hipMemcpy(stressTensor, dev_stressTensor, 9 * sizeof(float), hipMemcpyDeviceToHost);
    //outFile_stress << time << " " << stressTensor[0] << " " << stressTensor[1] << " " << stressTensor[2] << " " << stressTensor[3] << " " << stressTensor[4] << " " << stressTensor[5] << " " << stressTensor[6] << " " << stressTensor[7] << " " << stressTensor[8] << endl;

}  */