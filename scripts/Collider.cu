#include "hip/hip_runtime.h"
#include <cmath>
#include <omp.h>
#include <iostream>

#include "Collider.h"
#include "system.h"
#include "cuda_opt_constants.h"

// Collider methods
void Collider::Init(void) {
    potentialEnergy = 0;
}

// Opt version
__global__ void calculateForcesCUDA(Particle* particles, float* partialPotentialEnergy, int N, float Lx, float Ly, float Lz, float epsilon_sigma_6, float sigma_6, float cutoff) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float potentialEnergy = 0;
        float forceX = 0, forceY = 0, forceZ = 0;

        // Cache particle i's position in registers for faster access
        float posIX = particles[i].x;
        float posIY = particles[i].y;
        float posIZ = particles[i].z;

        for (int j = 0; j < N; j++) {
            if (i != j) {
                float dx = posIX - particles[j].x;
                float dy = posIY - particles[j].y;
                float dz = posIZ - particles[j].z;

                // Apply minimum image convention
                dx -= Lx * round(dx / Lx);
                dy -= Ly * round(dy / Ly);
                dz -= Lz * round(dz / Lz);

                float rsq = dx * dx + dy * dy + dz * dz;
                if (rsq < cutoff * cutoff) {
                    float d1 = 1.0 / rsq;
                    float d3 = d1 * d1 * d1;
                    float forceNormal = 24 * epsilon_sigma_6 * d3 * d1 * ( (2 * sigma_6 * d3) - 1);// - forceNormalCutOff_d * (cutoff / sqrt(rsq));

                    forceX += forceNormal * dx;
                    forceY += forceNormal * dy;
                    forceZ += forceNormal * dz;

                    //float r = sqrt(rsq);
                    //potentialEnergy += 4 * epsilon_sigma_6 * d3 * ( (sigma_6 * d3) - 1) - potentialEnergy_cutoff + forceCutoff * (r - cutoff);
                }

            }
        }
        particles[i].forceX = forceX;
        particles[i].forceY = forceY;
        particles[i].forceZ = forceZ;
        partialPotentialEnergy[i] = potentialEnergy;
    }
}

void Collider::CalculateForces(Particle* dev_particles, float* dev_partialPotentialEnergy, int N, float Lx, float Ly, float Lz) {
    //std::cout << "Epsilon: " << N << std::endl;

    // grid and block sizes
    int blockSize = 256; 
    int numBlocks = (N + blockSize - 1) / blockSize;

    // CUDA kernel launch
    calculateForcesCUDA<<<numBlocks, blockSize>>>(dev_particles, dev_partialPotentialEnergy, N, Lx, Ly, Lz, epsilon_sigma_6, sigma_6, cutoff);

    // errors
/*    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    }*/

    // synchronize device
    //hipDeviceSynchronize();
}


// Working version - Not optimal
/*__global__ void calculateForcesCUDA(Particle* particles, float* partialPotentialEnergy, int N, float Lx, float Ly, float Lz, float epsilon, float sigma, float cutoff) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float potentialEnergy = 0;

        particles[i].forceX = 0;
        particles[i].forceY = 0;
        particles[i].forceZ = 0;

        for (int j = 0; j < N; j++) {
            if (i != j) {
                float dx = particles[i].x - particles[j].x;
                float dy = particles[i].y - particles[j].y;
                float dz = particles[i].z - particles[j].z;

                dx -= Lx * round(dx / Lx);
                dy -= Ly * round(dy / Ly);
                dz -= Lz * round(dz / Lz);

                float distance = sqrt(dx * dx + dy * dy + dz * dz);
                
                if (distance < cutoff) { 
                    // Force at cutoff
                    float forceCutoff = -24*epsilon*((pow(sigma, 6) / pow(cutoff, 7)) - 2*(pow(sigma, 12) / pow(cutoff, 13))); 
                    float potentialEnergyAtCuttoff = 4*epsilon*(pow((sigma/cutoff), 12) - pow((sigma/cutoff), 6));

                    float forceNormal = -24*epsilon*((pow(sigma, 6) / pow(distance, 7)) - 2*(pow(sigma, 12) / pow(distance, 13))) - forceCutoff; 

                    particles[i].forceX += forceNormal * dx / distance;
                    particles[i].forceY += forceNormal * dy / distance;
                    particles[i].forceZ += forceNormal * dz / distance;

                    // Stoddard-Ford linear addition
                    potentialEnergy += 4*epsilon*(pow((sigma/distance), 12) - pow((sigma/distance), 6)) - potentialEnergyAtCuttoff + forceCutoff * (distance - cutoff);
                }
            }
        }
     partialPotentialEnergy[i] = potentialEnergy;
    }
}
*/